#include <stdio.h>
#include "helpers.cuh"


//Check whether RAM is full
void isMemoryFull(void *ptr){
	if (ptr==NULL){
		fprintf(stderr, "Memory Full.\nYour array is too large. Please try a smaller array.\n");
		exit(EXIT_FAILURE);
	}
}

//check whether file access is ok
void isFileOK(FILE *fp){
	if (fp==NULL){
		perror("A file access error occurred\n");
		exit(EXIT_FAILURE);
	}
}

//get the cuda error
char *getcudaError(hipError_t error){
    switch (error){

        case hipErrorMissingConfiguration:
            return "hipError_t Missing Configuration";

        case hipErrorOutOfMemory:
            return "hipError_t in Memory Allocation.. Device memory full. Try a smaller array.";

        case hipErrorNotInitialized:
            return "hipError_t Initialization Error";

        case hipErrorLaunchFailure:
            return "hipError_t Launch Failure";

        case hipErrorPriorLaunchFailure:
            return "hipError_t Prior Launch Failure";

        case hipErrorLaunchTimeOut:
            return "hipError_t Launch Timeout";

        case hipErrorLaunchOutOfResources:
            return "hipError_t Launch Out Of Resources";

        case hipErrorInvalidDeviceFunction:
            return "hipError_t Invalid Device Function";

        case hipErrorInvalidConfiguration:
            return "hipError_t Invalid Configuration";

        case hipErrorInvalidDevice:
            return "hipError_t Invalid Device";

        case hipErrorInvalidValue:
            return "hipError_t Invalid Value";

        case hipErrorInvalidPitchValue:
            return "hipError_t Invalid Pitch Value";

        case hipErrorInvalidSymbol:
            return "hipError_t Invalid Symbol";

        case hipErrorMapFailed:
            return "hipError_t Map Buffer Object Failed";

        case hipErrorUnmapFailed:
            return "hipError_t UnmapBuffer Object Failed";

        case cudaErrorInvalidHostPointer:
            return "hipError_t Invalid Host Pointer";

        case hipErrorInvalidDevicePointer:
            return "hipError_t Invalid Device Pointer";

        case hipErrorInvalidTexture:
            return "hipError_t Invalid Texture";

        case cudaErrorInvalidTextureBinding:
            return "hipError_t Invalid Texture Binding";

        case hipErrorInvalidChannelDescriptor:
            return "hipError_t Invalid Channel Descriptor";

        case hipErrorInvalidMemcpyDirection:
            return "hipError_t Invalid Memcpy Direction";

        case cudaErrorAddressOfConstant:
            return "hipError_t Address Of Constant";

        case cudaErrorTextureFetchFailed:
            return "hipError_t Texture Fetch Failed";

        case cudaErrorTextureNotBound:
            return "hipError_t Texture Not Bound";

        case cudaErrorSynchronizationError:
            return "hipError_t Synchronization Error";

        case cudaErrorInvalidFilterSetting:
            return "hipError_t Invalid Filter Setting";

        case cudaErrorInvalidNormSetting:
            return "hipError_t Invalid Norm Setting";

        case cudaErrorMixedDeviceExecution:
            return "hipError_t Mixed Device Execution";

        case hipErrorDeinitialized:
            return "hipError_t Cudart Unloading";

        case hipErrorUnknown:
            return "cuda Error Unknown";

        case cudaErrorNotYetImplemented:
            return "hipError_t Not Yet Implemented";

        case cudaErrorMemoryValueTooLarge:
            return "hipError_t Memory Value Too Large";

        case hipErrorInvalidHandle:
            return "hipError_t Invalid Resource Handle";

        case hipErrorNotReady:
            return "hipError_t Not Ready";

        case hipErrorInsufficientDriver:
            return "hipError_t Insufficient Driver";

        case hipErrorSetOnActiveProcess:
            return "hipError_t Set On Active Process";

        case cudaErrorInvalidSurface:
            return "hipError_t Invalid Surface";

        case hipErrorNoDevice:
            return "hipError_t No Device";

        case hipErrorECCNotCorrectable:
            return "hipError_t ECC Uncorrectable";

        case hipErrorSharedObjectSymbolNotFound:
            return "hipError_t Shared Object Symbol Not Found";

        case hipErrorSharedObjectInitFailed:
            return "cudaErrorS hared Object Init Failed";

        case hipErrorUnsupportedLimit:
            return "hipError_t Unsupported Limit";

        case cudaErrorDuplicateVariableName:
            return "hipError_t Duplicate Variable Name";

        case cudaErrorDuplicateTextureName:
            return "hipError_t Duplicate TextureName";

        case cudaErrorDuplicateSurfaceName:
            return "hipError_t Duplicate SurfaceName";

        case cudaErrorDevicesUnavailable:
            return "hipError_t Devices Unavailable";

        case hipErrorInvalidImage:
            return "hipError_t Invalid Kernel Image";

        case hipErrorNoBinaryForGpu:
            return "hipError_t NoKernel Image For Device";

        case cudaErrorIncompatibleDriverContext:
            return "hipError_t Incompatible Driver Context";

        case hipErrorPeerAccessAlreadyEnabled:
            return "hipError_t PeerAccess Already Enabled";

        case hipErrorPeerAccessNotEnabled:
            return "hipErrorPeerAccessNotEnabled";

        case hipErrorContextAlreadyInUse:
            return "hipError_t Device Already In Use";

        case hipErrorProfilerDisabled:
            return "hipErrorProfilerDisabled";

        case hipErrorProfilerNotInitialized:
            return "hipErrorProfilerNotInitialized";

        case hipErrorProfilerAlreadyStarted:
            return "hipErrorProfilerAlreadyStarted";

        case hipErrorProfilerAlreadyStopped:
            return "hipErrorProfilerAlreadyStopped";
		
		}
		return "<unknown>";
}
		
		
//check whether cuda errors
void checkCudaError(hipError_t status){
	if (status!=hipSuccess){
		fprintf(stderr,"Some Error occured in CUDA.\n:%s \nError Code : %d\n",getcudaError(status),status);
		exit(EXIT_FAILURE);
	}
}
