#include "hip/hip_runtime.h"


#include <stdio.h>
#include "helpers.cuh"
#include "data.cuh"
#include <string>
#include <hip/hip_runtime.h>

#define SAMPLES  20000    // samples * wavelength  must to be less than 50,000,00
#define WAVELENGTH 1024 // length of processed sampling points at a time
#define TOTAL  2048		// total samplings in a power trace  // 280 for lattice 1024 for giii
#define KEYBYTES 16
#define KEYS 256
#define CHUNK 2048	// number of chunk which must be loaded in to memory. chunk = wavelength *X ; x is an in

						//samples * CHUNK  must to be less than 4,000,000,000 this limit is from ram

__device__ byte hammingweight(byte M, byte R);
__device__ byte hamming(unsigned int *cipher, unsigned int i,unsigned int n,unsigned int key);
__global__ void maxCorelationkernel(double *corelation,double *wavestat,double *wavestat2,double *hammingstat);
__global__ void wavestatkernel(double *wavedata, double *wavestat,double *wavestat2,byte *hammingArray);
__global__ void hammingkernel(unsigned int *cipher,byte *hammingArray,double *hammingstat);

int main(int argc, char *argv[]){
//hipSetDevice(0);

//unsigned char dat;
float dat;
	
unsigned int i,j,k,temp;
//int l=atoi(argv[3]);
//char filename[80];
//strcpy(filename,"final");
//strcpy(filename,l+"");
//strcpy(filename,".txt");
FILE *file;
// empty the file. 
	file=fopen("all.txt","w");
	fprintf(file,"");
	fclose(file);
	
	double *MAXCorrelation=(double *)malloc(sizeof(double) * KEYS* KEYBYTES);
	isMemoryFull(MAXCorrelation);
	for (i=0;i<KEYS;i++){
		for(j=0;j<KEYBYTES;j++){
			MAXCorrelation[i*KEYBYTES+j]=0;
		}
	}
	
	//check args
	if(argc!=4){
		fprintf(stderr,"%s\n", "Not enough args. eg ./cpa wavedata.txt cipher.txt");
		exit(EXIT_FAILURE);
	}
	

	unsigned int *cipher=(unsigned int *)malloc(sizeof(unsigned int)*SAMPLES*KEYBYTES);
	isMemoryFull(cipher);

double *readWaveData=(double *)malloc(sizeof(double) * SAMPLES*CHUNK);
	isMemoryFull(readWaveData);
//space for corelation
	double *corelation=(double *)malloc(sizeof(double) * KEYS * KEYBYTES);
	isMemoryFull(corelation);
	
	
//get cipher texts
			
	file=fopen(argv[2],"r");
	isFileOK(file);
	for(i=0; i<SAMPLES ;i++){
		for(j=0; j<KEYBYTES; j++){
			fscanf(file,"%X",&cipher[(i/1)*KEYBYTES+j]);
			//fprintf(stderr,"%02x ",cipher[i*KEYBYTES+j]);
		}
		//fprintf(stderr,"%s","\n");
	}
	int numOfChunks=TOTAL/CHUNK;
	int l=0;
	for(l=0;l<numOfChunks;l++)
{	
		file=fopen(argv[1],"r");
		isFileOK(file);
		unsigned int rounds=TOTAL/WAVELENGTH;
		if(TOTAL%WAVELENGTH!=0) rounds++;
		int fileLength = strlen(argv[1]);
		char extention[4];
		strncpy(extention,argv[1]+fileLength-4,4) ;
		//if(strcmp(extention,"data") ==0 ){
		if(1){
	    fprintf(stderr,"%s\n",".data file is detected");
		//unsigned char  dat;
		
		//float dat;

/////////////////////	file read
	//fseek(file,sizeof(float)*CHUNK*l,SEEK_CUR);
		for(i=0; i<1*SAMPLES ;i++){
		   while(temp<CHUNK*l) {fread((void*)(&dat),sizeof(dat),1,file); temp++;}
		  
			temp=0;
			for(j=0; j<CHUNK; j++){
			fread((void*)(&dat),sizeof(dat),1,file);
                 	readWaveData[(i/1)*CHUNK+j]=(double)(dat);
					//fprintf(stderr,"%f ",dat);
					}
			//fseek(file,sizeof(float)*(TOTAL-CHUNK),SEEK_CUR);
			while(temp<TOTAL-(CHUNK*(l+1))) {fread((void*)(&dat),sizeof(dat),1,file); temp++;}
			temp=0;
				
			}
		}
		else{
		 long int dat;
			 fprintf(stderr,"%s\n",".txt file is detected");
			 for(i=0; i<SAMPLES ;i++){
	               	 for(j=0; j<WAVELENGTH; j++){
					fscanf(file,"%d",&dat); readWaveData[i*CHUNK+j]=(double)dat; 
					//printf("%ld ",dat);
							}
                        			}

			}	
	unsigned int insideRounds=CHUNK/WAVELENGTH;
	if(CHUNK%WAVELENGTH!=0) insideRounds++;
	for(k=0;k<insideRounds;k++) {   /// main loop
	
		//get wave data
	double *wavedata=(double *)malloc(sizeof(double) * SAMPLES*  WAVELENGTH);
	isMemoryFull(wavedata);
	fprintf(stderr,"%s %d %d \n","calculating", l,k);
    for(i=0; i<SAMPLES ;i++){
    for(j=0; j<WAVELENGTH; j++){
		double value1= readWaveData[i*CHUNK+j+k*WAVELENGTH];
		wavedata[i*WAVELENGTH+j] =value1;
	// memcpy(wavedata+(i*sizeof(double)), readWaveData+(i*sizeof(double)), sizeof(double));
	}
	}
	
	
	//Time
	
	//hipEvent_t start,stop;
	float elapsedtime;
	//hipEventCreate(&start);
	//hipEventRecord(start,0);
	//hipSetDevice(1);
	//cuda arrays and copying
	double *dev_wavedata;
	unsigned int *dev_cipher;
	double *dev_corelation,*dev_wavestat,*dev_wavestat2,*dev_hammingstat;
	byte *dev_hammingArray;
	hipMalloc((void**)&dev_wavedata, 1L*SAMPLES*WAVELENGTH*sizeof(double));
	hipMalloc((void**)&dev_cipher, 1L*SAMPLES*KEYBYTES*sizeof(unsigned int));
	hipMalloc((void**)&dev_corelation, KEYS*KEYBYTES*sizeof(double));
	hipMalloc((void**)&dev_hammingArray, 1L*KEYS*KEYBYTES*SAMPLES*sizeof(byte));
	hipMalloc((void**)&dev_wavestat, 2*WAVELENGTH*sizeof(double));
	hipMalloc((void**)&dev_wavestat2, 1L*KEYS*KEYBYTES*WAVELENGTH*sizeof(double));
	hipMalloc((void**)&dev_hammingstat, 2*KEYS*KEYBYTES*sizeof(double));
	
	hipMemcpy(dev_wavedata,wavedata,1L*SAMPLES*WAVELENGTH*sizeof(double),hipMemcpyHostToDevice);
	
	hipMemcpy(dev_cipher,cipher, 1L*SAMPLES*KEYBYTES*sizeof(unsigned int),hipMemcpyHostToDevice);

	dim3 grid(KEYBYTES/16,KEYS/16);
	dim3 block(16,16);

	//findhamming
	hammingkernel<<<grid,block>>>(dev_cipher,dev_hammingArray,dev_hammingstat);
	hipGetLastError();

	dim3 block3d(16,16,4);
	dim3 grid3d(KEYBYTES/16,KEYS/16,WAVELENGTH/4);
	//find wave stats
	wavestatkernel<<<grid3d,block3d>>>(dev_wavedata,dev_wavestat,dev_wavestat2,dev_hammingArray);
	hipGetLastError();

	//deploy double 
	maxCorelationkernel<<<grid,block>>>(dev_corelation,dev_wavestat,dev_wavestat2,dev_hammingstat);
	hipGetLastError();

	//copy back
	hipMemcpy(corelation,dev_corelation,KEYS*KEYBYTES*sizeof(double),hipMemcpyDeviceToHost);
	hipFree(dev_wavedata);
	hipFree(dev_cipher);
	hipFree(dev_corelation);
	hipFree(dev_wavestat);
	hipFree(dev_wavestat2);
	hipFree(dev_hammingstat);
	hipFree(dev_hammingArray);

	//Time
	//hipEventCreate(&stop);
	//hipEventRecord(stop,0);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&elapsedtime,start,stop);
	//fprintf(stderr,"Time spent for CUDA operation : %.10f\n",elapsedtime/(float)1000);

	//char finishtime[30];
	//time_t stopRound=time(NULL);
	//struct tm *loctime;
	//loctime = localtime (&stopRound);
	//strftime (finishtime, 30, "%Y-%m-%d %H:%M:%S", loctime);
	
	file=fopen("all.txt","a");
	fprintf(file,"\n");
	//fprintf(file,"%s \n",finishtime);
	fprintf(file,"%d,  pk0,  pk1,  pk2,  pk3,  pk4,  pk5,  pk6,  pk7,  pk8,  pk9, pk10, pk11, pk12, pk13, pk14, pk15, \n",SAMPLES);
	for (i=0;i<KEYS;i++){
	fprintf(file,"0x%02X,",i);
		for(j=0;j<KEYBYTES;j++){
			fprintf(file,"%.4f,",i,corelation[i*KEYBYTES+j]);
			double value1 =MAXCorrelation[i*KEYBYTES+j];
			double value2 = corelation[i*KEYBYTES+j];
			
			if(value1<value2) 
				MAXCorrelation[i*KEYBYTES+j] = value2;			
		}
		fprintf(file,"\n");
	}
	fprintf(file,"\n");
	fclose(file);

	free(wavedata);

	}

}
free(readWaveData);
double finalCorrelations[KEYS][KEYBYTES];
double n=0; int positions[KEYS][KEYBYTES];

	char finishtime[30];
 	time_t stop=time(NULL);
	//finishtime= ctime(&stop);  
	struct tm *loctime;
	loctime = localtime (&stop);
	strftime (finishtime, 30, "%Y-%m-%d %H:%M:%S", loctime);
	
	
	// write the csv file
	file=fopen("final.txt","a");
	fprintf(file,"\n");
	fprintf(file,"%s \n",finishtime);
	fprintf(file,"%d,  pk0,  pk1,  pk2,  pk3,  pk4,  pk5,  pk6,  pk7,  pk8,  pk9, pk10, pk11, pk12, pk13, pk14, pk15, \n",SAMPLES);
	for (i=0;i<KEYS;i++){
	fprintf(file,"0x%02X,",i);
		for(j=0;j<KEYBYTES;j++){
			fprintf(file,"%.4f,",MAXCorrelation[i*KEYBYTES+j]);
			finalCorrelations[i][j]=MAXCorrelation[i*KEYBYTES+j];
			}
		fprintf(file,"\n");
	}
////////////////////////////// sort the results /// 
	int p=0;
	for(j=0;j<KEYBYTES;j++){
		for(i=0;i<KEYS;i++) positions[i][j] =i;
		for (p=0;p<255;p++){
		
		for (i=0;i<KEYS-p-1;i++){
					if(finalCorrelations[i][j]<finalCorrelations[i+1][j]) { 
						n=finalCorrelations[i][j];
						finalCorrelations[i][j]=finalCorrelations[i+1][j];
						finalCorrelations[i+1][j]=n; 
					
						n=positions[i][j];
						positions[i][j]=positions[i+1][j];
						positions[i+1][j]=n; 
						}
					
					}
			}
	}
	
	for(j=0;j<KEYBYTES;j++){
	printf("  |%d|\t",j);
	}
	printf("\n");
	
	for (i=0;i<5;i++){
	for(j=0;j<KEYBYTES;j++){
		printf("  %02x\t",positions[i][j]);
		}
		printf("\n");
	for(j=0;j<KEYBYTES;j++){
		printf("%.4f \t",finalCorrelations[i][j]);
		}
		printf("\n\n");
	}
		
	
	
	return 0;
}


__device__ byte hammingweight(byte M, byte R){
	byte H=M^R;
	// Count the number of set bits
	byte dist=0;
	while(H){
		dist++; 
		H &= H - 1;
}
	

	return dist;
}

__device__ byte hamming(unsigned int *cipher, unsigned int i,unsigned int n,unsigned int key) { //n is byteno  i is the sample
      byte st10 = (byte)cipher[i*KEYBYTES+inv_shift[n]];
      byte st9 = (byte)inv_sbox[cipher[i*KEYBYTES+n]  ^ key] ;
      byte st19 = (byte)(inv_sbox[cipher[i*KEYBYTES+inv_shift[n] ]  ^ key]) ;

//   byte st9 = (byte)((tbox[cipher[i*KEYBYTES+n]  ^ key]>>24));
//   byte st9 = (byte)(inv_sbox[cipher[i*KEYBYTES+n]  ^ key ]);
//  byte st10 = (byte)(cipher[i*KEYBYTES+n] ^key );

//	byte dist=0;
//if((st9 ^ st10) & 0x0FF)
//idf((st9 ^ st10) & 0x0)
//		 dist= 1;
//	else
//		 dist= 0;  

   byte dist = hammingweight(st9,st10);
 

// byte dist = hammingweight(st9&0xf0,st10&0xf0);	  
//   byte dist = hammingweight(st9&0x01,0);
// byte dist = hammingweight(st19 ,0);

//  byte dist = hammingweight(st9, 0);
//

	return dist;
}


__global__ void maxCorelationkernel(double *corelation,double *wavestat,double *wavestat2,double *hammingstat){
	
	int keyguess=blockDim.y*blockIdx.y+threadIdx.y;
	int keybyte=blockDim.x*blockIdx.x+threadIdx.x;

	if (keybyte<KEYBYTES && keyguess<KEYS ){ 
		
		double sigmaH,sigmaH2,sigmaW=0,sigmaW2=0,sigmaWH=0;	
		sigmaH=hammingstat[KEYBYTES*keyguess+keybyte];
		sigmaH2=hammingstat[KEYS*KEYBYTES+KEYBYTES*keyguess+keybyte];
		double temp_corelation=0;;
		double corelationmax=0;;
		unsigned int j;
		for(j=0;j<WAVELENGTH;j++){
			
			sigmaWH=wavestat2[j*KEYS*KEYBYTES + keyguess*KEYBYTES + keybyte];
			sigmaW=wavestat[j];
			sigmaW2=wavestat[WAVELENGTH+j];

			double numerator=SAMPLES*sigmaWH - sigmaW*sigmaH;
			double denominator=sqrt(SAMPLES*sigmaW2 - sigmaW*sigmaW)*sqrt(SAMPLES*sigmaH2 - sigmaH*sigmaH);
			temp_corelation=fabs(numerator/denominator);
		
			if(temp_corelation>corelationmax){
				corelationmax=temp_corelation;
			}
		}

		corelation[keyguess*KEYBYTES+keybyte]=corelationmax;
	}
	return;
}

__global__ void wavestatkernel(double *wavedata, double *wavestat,double *wavestat2,byte *hammingArray){

	int keyguess=blockDim.y*blockIdx.y+threadIdx.y;
	int keybyte=blockDim.x*blockIdx.x+threadIdx.x;
	int wave=blockDim.z*blockIdx.z+threadIdx.z;

	if (keyguess<KEYS && keybyte<KEYBYTES && wave<WAVELENGTH ){
		unsigned int i;
		double sigmaWH=0;
		for(i=0;i<SAMPLES;i++){
			sigmaWH+=wavedata[i*WAVELENGTH+wave]*(double)hammingArray[i*KEYS*KEYBYTES + keyguess*KEYBYTES + keybyte];
		}
		wavestat2[wave*KEYS*KEYBYTES + keyguess*KEYBYTES + keybyte ]=sigmaWH;
	}

	if (keyguess==0 && keybyte==0 && wave<WAVELENGTH ){
		unsigned int i;
		double sigmaW=0,sigmaW2=0,W=0;
		for(i=0;i<SAMPLES;i++){
			W=wavedata[i*WAVELENGTH+wave];
			sigmaW+=W;
			sigmaW2+=W*W;
		}
		wavestat[wave]=sigmaW;
		wavestat[WAVELENGTH+wave]=sigmaW2;
	}
	return;
}

__global__ void hammingkernel(unsigned int *cipher,byte *hammingArray,double *hammingstat){
	int keyguess=blockDim.y*blockIdx.y+threadIdx.y;
	int keybyte=blockDim.x*blockIdx.x+threadIdx.x;

	if (keybyte<KEYBYTES && keyguess<KEYS ){
		double sigmaH=0,sigmaH2=0;
		byte H;
		unsigned int i;
		for(i=0;i<SAMPLES;i++){
			H=hamming(cipher,i,keybyte,keyguess);
			hammingArray[i*KEYS*KEYBYTES + keyguess*KEYBYTES + keybyte]=H;
			sigmaH+=(double)H;
			sigmaH2+=(double)H*(double)H;
		}
		hammingstat[KEYBYTES*keyguess+keybyte]=sigmaH;
		hammingstat[KEYS*KEYBYTES+KEYBYTES*keyguess+keybyte]=sigmaH2;
	}
	return;
}

